#include "hip/hip_runtime.h"
/*
 * Copyright 2014 Google Inc. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *    http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include "../include/util.cuh"
#include "../include/worker.cuh"
#include "../include/timer.cuh"

using namespace std;

/* 
 * ====================
 * WorkResult
 * ====================
 */
WorkResult::WorkResult(WorkResult::RESULTS resultType, Cost& results) : _resultType(resultType), _results(&results) {
}

WorkResult::WorkResult(WorkResult::RESULTS resultType) : _resultType(resultType), _results(NULL) {
}

WorkResult::~WorkResult() {
    delete _results; // delete NULL is ok
}

Cost& WorkResult::getResults() const {
    return *_results;
}

WorkResult::RESULTS WorkResult::getResultType() const {
    return _resultType;
}

/* 
 * ====================
 * Worker
 * ====================
 */
Worker::Worker(ConvNet& convNet) : _convNet(&convNet) {
}

Worker::~Worker() {
}

/* 
 * ====================
 * DataWorker
 * ====================
 */
DataWorker::DataWorker(ConvNet& convNet, CPUData& data) : Worker(convNet), _data(&data) {
    _dp = &convNet.getDataProvider();
    assert(_data != NULL);
    _dp->setData(*_data);
}

DataWorker::~DataWorker() {
    _dp->clearData();
}

/* 
 * ====================
 * TrainingWorker
 * ====================
 */
TrainingWorker::TrainingWorker(ConvNet& convNet, CPUData& data, double progress, bool test)
    : DataWorker(convNet, data), _progress(progress), _test(test) {
}

bool TrainingWorker::run() {
	_convNet->setTrainingProgress(_progress);
    Cost& batchCost = *new Cost();
//    Timer t, t2;
//    t2.start();
    int numMinibatches = _dp->getNumMinibatches();
    for (int i = 0; i < numMinibatches; i++) {
        Timer miniTimer;
//        miniTimer.start();
        for (int p = 0; p < _convNet->getNumPasses(); p++) {
//            t.start();
            _convNet->fprop(i, p, _test ? PASS_TEST : PASS_TRAIN);
//            printf("Worker: mini=%d, pass=%d fprop took %.2fmsec\n", i, p, t.stop());
//            t.start();
            _convNet->getCost(batchCost);
//            printf("Worker: mini=%d, pass=%d getcost took %.2fmsec\n", i, p, t.stop());

            if (!_test) {
//                t.start();
                _convNet->bprop(p, PASS_TRAIN);
//                printf("Worker: mini=%d, pass=%d bprop took %.2fmsec\n", i, p, t.stop());
//                t.start();
                _convNet->updateWeights(p);
//                printf("Worker: mini=%d, pass=%d updateWeights took %.2fmsec\n", i, p, t.stop());
            }
        }
//        printf("Worker: minibatch %d took %.2fmsec\n", i, miniTimer.stop());
    }
//    printf("Worker: batch took %.2fmsec\n", t2.stop());
    _convNet->getResultQueue().enqueue(new WorkResult(WorkResult::BATCH_DONE, batchCost));
    return false;
}

/*
 * ====================
 * SyncWorker
 * ====================
 */
SyncWorker::SyncWorker(ConvNet& convNet) : Worker(convNet) {
}

bool SyncWorker::run() {
    _convNet->copyToCPU();
    _convNet->getResultQueue().enqueue(new WorkResult(WorkResult::SYNC_DONE));
    return false;
}

/*
 * ====================
 * ExitWorker
 * ====================
 */
ExitWorker::ExitWorker(ConvNet& convNet) : Worker(convNet) {
}

bool ExitWorker::run() {
    return true;
}

/* 
 * ====================
 * GradCheckWorker
 * ====================
 */
GradCheckWorker::GradCheckWorker(ConvNet& convNet, CPUData& data) 
    : DataWorker(convNet, data) {
}

bool GradCheckWorker::run() {
    _convNet->checkGradients();
    exit(0); // eh
    return true;
//    exit(0);
}

/* 
 * ====================
 * MultiviewTestWorker
 * ====================
 */
MultiviewTestWorker::MultiviewTestWorker(ConvNet& convNet, CPUData& data, int numViews, Matrix& cpuProbs, const char* logregName) 
    : DataWorker(convNet, data), _numViews(numViews), _cpuProbs(&cpuProbs), _logregName(logregName) {
//    assert(_data->getNumCases() % _numViews == 0);
//    assert(convNet.getNumReplicas() == 1); // For now?
}

MultiviewTestWorker::MultiviewTestWorker(ConvNet& convNet, CPUData& data, int numViews) 
    : DataWorker(convNet, data), _numViews(numViews), _cpuProbs(NULL), _logregName("") {
//    assert(_data->getNumCases() % _numViews == 0);
}

MultiviewTestWorker::~MultiviewTestWorker() {
//    delete _cpuProbs;
}

CPUData& MultiviewTestWorker::getMinibatch(int v, int i) {
    int numCasesPerView = _dp->getNumCases() / _numViews;
    int miniStart = v * numCasesPerView + i * _dp->getMinibatchSize();
    int miniEnd = v * numCasesPerView + min(numCasesPerView, (i + 1) * _dp->getMinibatchSize());
    CPUData& mini = _dp->getDataSlice(miniStart, miniEnd);
    return mini;
}

bool MultiviewTestWorker::run() {
    int numCasesPerView = _dp->getNumCases() / _numViews;
    int numMiniPerView = DIVUP(numCasesPerView, _dp->getMinibatchSize());

    Cost& batchCost = *new Cost();
    for (int i = 0; i < numMiniPerView; i++) {
        for (int v = 0; v < _numViews - 1; v++) {
            for (int p = 0; p < _convNet->getNumPasses(); p++) {
                _convNet->fprop(getMinibatch(v, i), p, v == 0 ? PASS_MULTIVIEW_TEST_START : PASS_MULTIVIEW_TEST);
            }
        }
        for (int p = 0; p < _convNet->getNumPasses(); p++) {
            _convNet->fprop(getMinibatch(_numViews - 1, i), p, PASS_MULTIVIEW_TEST_END);
            _convNet->getCost(batchCost);
        }
//        if (_cpuProbs != NULL) {
//            LogregCostLayer& logregLayer = *dynamic_cast<LogregCostLayer*>(&_convNet->getLayer(_logregName, 0));
//            NVMatrix::setDeviceID(logregLayer.getDeviceID());
//            Matrix& miniProbs = _cpuProbs->sliceRows(i * _dp->getMinibatchSize(),
//                                                     min(numCasesReal, (i + 1) * _dp->getMinibatchSize()));
//            NVMatrix& acts = logregLayer.getProbsAccum();
//            NVMatrix acts_T;
//            acts.transpose(acts_T);
//            acts_T.copyToHost(miniProbs);
//
//            delete &miniProbs;
//        }
    }
    _convNet->getResultQueue().enqueue(new WorkResult(WorkResult::BATCH_DONE, batchCost));
    return false;
}

/* 
 * ====================
 * FeatureWorker
 * ====================
 */
FeatureWorker::FeatureWorker(ConvNet& convNet, CPUData& data, MatrixV& ftrs, stringv& layerNames, bool deleteFeatures)
    : DataWorker(convNet, data), _ftrs(&ftrs), _layerNames(&layerNames), _deleteFeatures(deleteFeatures) {
    assert(layerNames.size() == ftrs.size());
    for (int i = 0; i < layerNames.size(); i++) {
        assert(ftrs[i]->getNumRows() == data.getNumCases());
        assert(!ftrs[i]->isTrans());
    }
}

FeatureWorker::~FeatureWorker() {
    if (_deleteFeatures) {
        for (int i = 0; i < _ftrs->size(); i++) {
            delete _ftrs->at(i);
        }
        delete _ftrs;
    }
    delete _layerNames;
}

bool FeatureWorker::run() {
    Cost& batchCost = *new Cost();
    map<int,int> repStart; // Feature write start offsets within minibatch
    for (int i = 0; i < _dp->getNumMinibatches(); i++) {
        for (int f = 0; f < _layerNames->size(); f++) {
            repStart[f] = 0;
        }

        for (int p = 0; p < _convNet->getNumPasses(); p++) {
            _convNet->fprop(i, p, PASS_FEATURE_GEN);
            _convNet->getCost(batchCost);
            for (int f = 0; f < _layerNames->size(); f++) {

                if (_convNet->getLayer(_layerNames->at(f), 0).getFwdActiveInputReplicaIdx(p) >= 0) {
                    Matrix& miniFtrs = _ftrs->at(f)->sliceRows(i * _dp->getMinibatchSize(),
                                                               min(_dp->getNumCases(), (i + 1) * _dp->getMinibatchSize()));

                    for (int r = 0; r < _convNet->getLayer(_layerNames->at(f), 0).getNumReplicas(); ++r) {
                        Layer& ftrLayer = _convNet->getLayer(_layerNames->at(f), r);
                        int d = ftrLayer.getDeviceID();
                        NVMatrix::setDeviceID(d);
                        NVMatrix& acts = ftrLayer.getActs();

                        Matrix& repMiniFtrs = miniFtrs.sliceRows(repStart[f],
                                                                 min(int(miniFtrs.getNumRows()), repStart[f] + acts.getLeadingDim()));

                        NVMatrix acts_T;
                        acts.transpose(false);
                        acts.transpose(acts_T);
                        acts_T.copyToHost(repMiniFtrs);
                        NVMatrix::syncStream(); // eh why not

                        delete &repMiniFtrs;

                        repStart[f] += acts.getLeadingDim();
                    }
                    delete &miniFtrs;
                }
            }
        }
    }
    _convNet->getResultQueue().enqueue(new WorkResult(WorkResult::BATCH_DONE, batchCost));
    return false;
}

/* 
 * ====================
 * DataGradWorker
 * ====================
 */
DataGradWorker::DataGradWorker(ConvNet& convNet, CPUData& data, Matrix& dataGrads, int dataLayerIdx, int softmaxLayerIdx)
    : DataWorker(convNet, data), _dataGrads(&dataGrads), _dataLayerIdx(dataLayerIdx), _softmaxLayerIdx(softmaxLayerIdx) {
//    assert(dataGrads.getNumRows() == data.getNumCases());
//    assert(!dataGrads.isTrans());
}

DataGradWorker::~DataGradWorker() {
//    delete _dataGrads;
}

bool DataGradWorker::run() {
//    DataLayer& dataLayer = *dynamic_cast<DataLayer*>(&_convNet->getLayer(_dataLayerIdx));
//    SoftmaxLayer& softmaxLayer = *dynamic_cast<SoftmaxLayer*>(&_convNet->getLayer(_softmaxLayerIdx));
//    softmaxLayer.setDoLogregGrad(false);
//    Cost& batchCost = *new Cost(0);
//    for (int i = 0; i < _dp->getNumMinibatches(); i++) {
//        _convNet->fprop(i, PASS_TEST);
//        _convNet->getCost(batchCost);
//        softmaxLayer.getActs().apply(NVMatrixOps::Log(), softmaxLayer.getActsGrad());
//        
//        softmaxLayer.getActsGrad().addScalar(1);
//        softmaxLayer.getActsGrad().scale(-1);
//        softmaxLayer.incRcvdBInputs();
//        softmaxLayer.bprop(PASS_TEST);
//        
//        Matrix& miniDataGrads = _dataGrads->sliceRows(i * _dp->getMinibatchSize(),
//                                                      min(_dp->getNumCases(), (i + 1) * _dp->getMinibatchSize()));
//        NVMatrix& grads = dataLayer.getActsGrad();
//        NVMatrix grads_T;
//        if (grads.isTrans()) {
//            NVMatrix& soft_T = grads.getTranspose();
//            soft_T.transpose(grads_T);
//            delete &soft_T;
//        } else {
//            grads.transpose(grads_T);
//        }
//        grads_T.copyToHost(miniDataGrads);
//        delete &miniDataGrads;
//        
//        _convNet->reset();
//    }
//    hipDeviceSynchronize();
//    _convNet->getResultQueue().enqueue(new WorkResult(WorkResult::BATCH_DONE, batchCost));
    return false;
}
